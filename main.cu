//////////////////////////////////////////////////////////////
// Authors  : Lama Afra, Mohammad Al Khalidi, Taysseer Samman
// Usernames: laa59, mwa30, tjs00
// Course   : CMPS 396AA
// Timestamp: 20200328
// Project  : Parallel Quicksort
/////////////////////////////////////////////////////////////

#include "common.h"
#include "timer.h"

// CUDA Windows Headers
#if defined _WIN32 || defined _WIN64
#include "hip/hip_runtime.h"
#include ""
#endif

// Swap two elements of an array
void swap_cpu(float* a, float* b)
{
	float temp = *a;
	*a = *b;
	*b = temp;
}

// Computes the partition after rearranging the array
int partition_cpu(float* arr, int arrSize)
{
	// Index of smaller element
    int i = - 1;

	for (int j = 0; j < arrSize - 1; j++)
	{
		// If current element is smaller than the pivot
		if (arr[j] < arr[arrSize - 1])
		{
			// Increment the index of the smaller element
			i++;
			// Swap array elements with indices i and j
			swap_cpu(&arr[i], &arr[j]);
		}
	}

	// Swap array elements with indices i + 1 and pivot
	swap_cpu(&arr[i + 1], &arr[arrSize - 1]);

	// Return parition index
    return (i + 1);
}

// Sorts an array with the quick sort algorithm
void quicksort_cpu(float* arr, int arrSize)
{
	// Array size must be greater than 1
	if (arrSize > 1)
	{
		// Partition
        int k = partition_cpu(arr, arrSize);

		// Sort the left partition
		quicksort_cpu(&arr[0], k);

		// Sort the right partition
		quicksort_cpu(&arr[k + 1], arrSize - k - 1);
	}
}

int main(int argc, char**argv)
{
    hipDeviceSynchronize();

    // Allocate memory and initialize data
    Timer timer;
    unsigned int arrSize = (argc > 1)?(atoi(argv[1])):ARRAY_SIZE;
    float* arr_cpu = (float*) malloc(arrSize * sizeof(float));
    float* arr_gpu = (float*) malloc(arrSize * sizeof(float));

    //Global array which will be used by the partition kernel
    float* arrCopy_gpu = (float*) malloc(arrSize * sizeof(float));
    float* lessThan_gpu = (float*) malloc(arrSize * sizeof(float));
    float* greaterThan_gpu = (float*) malloc(arrSize * sizeof(float));
    float* partition_gpu = (float*) malloc(arrSize * sizeof(float));
    
	for (unsigned int i = 0; i < arrSize; ++i) 
	{
        float val = rand();
        arr_cpu[i] = val;
        arr_gpu[i] = val;
    }

    // Compute on CPU
    startTime(&timer);
	quicksort_cpu(arr_cpu, arrSize);
    stopTime(&timer);
    printElapsedTime(timer, "CPU time");

    // Compute on GPU
    startTime(&timer);
	quicksort_gpu(arr_gpu, arrSize); //arrCopy_gpu, lessThan_gpu, greaterThan_gpu
    stopTime(&timer);
    printElapsedTime(timer, "GPU time");

    printf("\n");
    for (unsigned int i = 0; i < arrSize; ++i) {
        printf("%e ", arr_cpu[i]);
    }
    printf("\n");

    printf("\n");
    for (unsigned int i = 0; i < arrSize; ++i) {
        printf("%e ", arr_gpu[i]);
    }
    printf("\n");

    // Verify result
    for(unsigned int i = 0; i < arrSize; ++i) 
	{
        if(arr_cpu[i] != arr_gpu[i])
        {
            printf("Mismatch at index %u (CPU result = %e, GPU result = %e)\n", i, arr_cpu[i], arr_gpu[i]);
            exit(0);
        }
    }

    // Free memory
    free(arr_cpu);
    free(arr_gpu);

    //Exit program
    return 0;
}

