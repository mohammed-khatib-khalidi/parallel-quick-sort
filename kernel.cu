#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////
// Authors  : Lama Afra, Mohammed Al Khalidi, Taysseer Samman
// Usernames: laa59, mwa30, tjs00
// Course   : CMPS 396AA
// Timestamp: 20200328
// Project  : Parallel Quicksort
/////////////////////////////////////////////////////////////

#include "common.h"
#include "timer.h"

// CUDA Windows Headers
#if defined _WIN32 || defined _WIN64
#include "hip/hip_runtime.h"
#include ""
#endif

#define BLOCK_DIM 1024


// Swap two elements of an array
__device__ void swap_gpu(int* a, int* b)
{
	int temp = *a;
	*a = *b;
	*b = temp;
}

// A sequential version of the selection sort
// This algorithm will be applied after reaching the maximum recursion depth on gpu
__device__ void selectionSort(int* arr, int arrSize) 
{ 
    int i, j, min_idx; 
  
    // One by one move boundary of unsorted subarray 
    for (i = 0; i < arrSize - 1; i++) 
    { 
        // Find the minimum element in unsorted array 
        min_idx = i; 
        for (j = i+1; j < arrSize; j++)
        {
            if (arr[j] < arr[min_idx])
            {
                min_idx = j;
            }
        }
  
        // Swap the found minimum element with the first element 
        swap_gpu(&arr[min_idx], &arr[i]); 
    } 
}

// Computes the partition after rearranging the array
__device__ int partition_gpu(int* arr, int arrSize)
{
	// Index of smaller element
    int i = - 1;

	for (int j = 0; j < arrSize - 1; j++)
	{
		// If current element is smaller than the pivot
		if (arr[j] < arr[arrSize - 1])
		{
			// Increment the index of the smaller element
			i++;
			// Swap array elements with indices i and j
			swap_gpu(&arr[i], &arr[j]);
		}
	}

	// Swap array elements with indices i + 1 and pivot
	swap_gpu(&arr[i + 1], &arr[arrSize - 1]);

	// Return partition index
    return (i + 1);
}

// Naive version of the parallel quicksort which only parallelizes recursive calls
__global__ void quicksort_naive_kernel(int* arr, int arrSize, int depth)
{
    // If depth is more than maximum recursion
    // Apply sequential selection sort
    if(depth > MAX_RECURSION)
    {
        selectionSort(arr, arrSize);
        return;
    }

    // Partition
    int k = partition_gpu(arr, arrSize);

    if(k > 1) 
	{
        // Create cuda stream to run recursive calls in parallel
        hipStream_t s_left;

        // Set the non-blocking flag for the cuda stream
        hipStreamCreateWithFlags(&s_left, hipStreamNonBlocking);

        // Sort the left partition
		quicksort_naive_kernel <<< 1, 1, 0, s_left >>> (&arr[0], k, depth + 1);

        // Destroy the stream after getting done from it
        hipStreamDestroy(s_left);
    }

    if(arrSize > k + 2) 
	{
        // Create cuda stream to run recursive calls in parallel
        hipStream_t s_right;

        // Set the non-blocking flag for the cuda stream
        hipStreamCreateWithFlags(&s_right, hipStreamNonBlocking);

        // Sort the right partition
		quicksort_naive_kernel <<< 1, 1, 0, s_right >>> (&arr[k + 1], arrSize - k - 1, depth + 1);

        // Destroy the stream after getting done from it
        hipStreamDestroy(s_right);
    }
}

// The partition kernel method
// The array size should be usually double the number of threads since each thread will be responsible for two array elements
__global__ void partition_kernel (
    int* arr,
    int* arrCopy,
    int* lessThan,
    int* greaterThan,
    int* lessThanSums,
    int* greaterThanSums,
    int* partitionArr,
    int* blockCounter,
    int* flags,
    int arrSize)
{
    // Shared memory
    __shared__ int bid_s;
    __shared__ int lsPrevSum_s;
    __shared__ int gtPrevSum_s;
    __shared__ int lsLocalSum_s;
    __shared__ int gtLocalSum_s;
    __shared__ int lessThan_s[2 * BLOCK_DIM];
    __shared__ int greaterThan_s[2 * BLOCK_DIM];

    // If this was the first thread
    if (threadIdx.x == 0)
    {
        //Get current block index and increment by 1
        bid_s = atomicAdd(&blockCounter[0], 1);
    }

    // Synchronize all threads
    __syncthreads();

    //Get the dynamic block id
    const int bid = bid_s;    

    // Load the real thread position
    int i = (2 * blockDim.x * bid) + threadIdx.x;

    // ========================= Copy to temporary, lessThan and greaterThan arrays =========================

    // Choose the middle element as the pivot
    int pivot = arr[(arrSize - 1) / 2];

    // Handle first element by the thread
    if(i < arrSize)
    {
        // Copy to temporary array
        arrCopy[i] = arr[i];

        // Copy to the lessThan array
        if(arr[i] < pivot)
        {
            lessThan_s[i] = 1;
        }
        else
        {
            lessThan_s[i] = 0;
        }

        // Copy to the greaterThan array
        if(arr[i] > pivot)
        {
            greaterThan_s[i] = 1;
        }
        else
        {
            greaterThan_s[i] = 0;
        }
    }

    // Handle second element by the thread
    if(i + blockDim.x < arrSize)
    {
        arrCopy[i + blockDim.x] = arr[i + blockDim.x];

        // Copy to the lessThan array
        if(arr[i + blockDim.x] < pivot)
        {
            lessThan_s[i + blockDim.x] = 1;
        }
        else
        {
            lessThan_s[i + blockDim.x] = 0;
        }

        // Copy to the greaterThan array
        if(arr[i + blockDim.x] > pivot)
        {
            greaterThan_s[i + blockDim.x] = 1;
        }
        else
        {
            greaterThan_s[i + blockDim.x] = 0;
        }
    }

    // ========================= Prefix sum (lessThan & greaterThan) =========================

    // ========================= Reduction phase =========================

    for(unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {       
        // Synchronize all threads
        __syncthreads();
        // Re-index threads to minimize divergence
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if(index >= stride && index < 2 * blockDim.x) {
            lessThan_s[index] += lessThan_s[index - stride];
            greaterThan_s[index] += greaterThan_s[index - stride];
        }
    }

    // ========================= Post-Reduction phase =========================

    for (int stride = BLOCK_DIM / 2; stride > 0; stride /= 2)
    {
        // Synchronize all threads
        __syncthreads();
        // Re-index threads to minimize divergence
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if(index + stride < 2 * blockDim.x)
        {
            lessThan_s[index + stride] += lessThan_s[index];
            greaterThan_s[index + stride] += greaterThan_s[index];
        }
    }

    // ========================= Write partial sums =========================

    // Synchronize all threads
    __syncthreads();

    // If this was the last thread
    if (threadIdx.x == blockDim.x - 1)
    {
        lsLocalSum_s = lessThan_s[2 * BLOCK_DIM - 1];
        gtLocalSum_s = greaterThan_s[2 * BLOCK_DIM - 1];
    }

    // ========================= Single pass scan =========================

    // If this was the first thread
    if (threadIdx.x == 0)
    {
        // Wait for previous flag
        while (atomicAdd(&flags[bid], 0) == 0){;}
        
        // Check if there are blocks before
        if(bid > 0)
        {
            // Read previous partial sums
            lsPrevSum_s = lessThanSums[bid];
            gtPrevSum_s = greaterThanSums[bid];
        }
        else
        {
            // No previous sums, set to zero
            lsPrevSum_s = 0.0f;
            gtPrevSum_s = 0.0f;
        }

        // Propagate partial sum
        lessThanSums[bid + 1] = lsPrevSum_s + lsLocalSum_s;
        greaterThanSums[bid + 1] = gtPrevSum_s + gtLocalSum_s;

        // Memory fence
        __threadfence();

        // Set flag
        atomicAdd(&flags[bid + 1], 1);
    }

    // Synchronize all threads
    __syncthreads();

    // ========================= Commit changes to global memory =========================

    if (i < arrSize)
    {
        lessThan[i] = lessThan_s[threadIdx.x] + lsPrevSum_s + lsLocalSum_s;
        greaterThan[i] = greaterThan_s[threadIdx.x] + gtPrevSum_s + gtLocalSum_s;
    }

    if (i + blockDim.x < arrSize)
    {
        lessThan[i + blockDim.x] = lessThan_s[threadIdx.x + blockDim.x] + lsPrevSum_s + lsLocalSum_s;
        greaterThan[i + blockDim.x] = greaterThan_s[threadIdx.x + blockDim.x] + gtPrevSum_s + gtLocalSum_s;
    }
}

//Advanced version of the parallel quicksort which parallelizes both the partition method and the recursive calls
__global__ void quicksort_advanced_kernel(
    int* arr,
    int* arrCopy,
    int* lessThan,
    int* greaterThan,
    int* lessThanSums,
    int* greaterThanSums,
    int* partitionArr,
    int* blockCounter,
    int* flags,
    int depth,
    int arrSize)
{
    // If depth is more than maximum recursion
    // Apply sequential selection sort
    if(depth > MAX_RECURSION)
    {
        selectionSort(arr, arrSize);
        return;
    }

    // Configure the number of blocks and threads per block
    const unsigned int numThreadsPerBlock = BLOCK_DIM;
    const unsigned int numElementsPerBlock = 2 * numThreadsPerBlock;
    const unsigned int numBlocks = (arrSize + numElementsPerBlock - 1)/numElementsPerBlock;

	// Partition
    partition_kernel <<< numBlocks, numThreadsPerBlock >>> (arr, arrCopy, lessThan, greaterThan, lessThanSums, greaterThanSums, partitionArr, blockCounter, flags, arrSize);

    // Set partition as first element of the array after the partition kernel has done its work
    int k = partitionArr[0];

    if(k > 1)
	{
        // Create cuda stream to run recursive calls in parallel
        hipStream_t s_left;

        // Set the non-blocking flag for the cuda stream
        hipStreamCreateWithFlags(&s_left, hipStreamNonBlocking);

        // Sort the left partition
		quicksort_advanced_kernel <<< 1, 1, 0, s_left >>> (
            &arr[0],
            &arrCopy[0],
            &lessThan[0],
            &greaterThan[0],
            &lessThanSums[0],
            &greaterThanSums[0],
            &partitionArr[0],
            &blockCounter[0],
            &flags[0],
            depth + 1,
            k
        );

        // Destroy the stream after getting done from it
        hipStreamDestroy(s_left);
    }

    if(arrSize > k + 2) 
	{
        // Create cuda stream to run recursive calls in parallel
        hipStream_t s_right;

        // Set the non-blocking flag for the cuda stream
        hipStreamCreateWithFlags(&s_right, hipStreamNonBlocking);

        // Sort the right partition
		quicksort_advanced_kernel <<< 1, 1, 0, s_right >>> (
            &arr[k + 1],
            &arrCopy[k + 1],
            &lessThan[k + 1],
            &greaterThan[k + 1],
            &lessThanSums[k + 1],
            &greaterThanSums[k + 1],
            &partitionArr[k + 1],
            &blockCounter[k + 1],
            &flags[k + 1],
            depth + 1,
            arrSize - k - 1
        );

        // Destroy the stream after getting done from it
        hipStreamDestroy(s_right);
    }
}

__host__ void quicksort_gpu(int* arr, int arrSize)
{
    //Define the timer
    Timer timer;

    //Allocate GPU memory
    startTime(&timer);
    
    //Declare and allocate required arrays on the device
    int* arr_d;
    int* arrCopy;
    int* lessThan;
    int* greaterThan;
    int* lessThanSums;
    int* greaterThanSums;
    int* partitionArr;
    int* blockCounter;
    int* flags;

    hipMalloc((void**) &arr_d, arrSize * sizeof(int));
    hipMalloc((void**) &arrCopy, arrSize * sizeof(int));
    hipMalloc((void**) &lessThan, arrSize * sizeof(int));
    hipMalloc((void**) &greaterThan, arrSize * sizeof(int));
    hipMalloc((void**) &lessThanSums, arrSize * sizeof(int));
    hipMalloc((void**) &greaterThanSums, arrSize * sizeof(int));
    hipMalloc((void**) &partitionArr, arrSize * sizeof(int));
    hipMalloc((void**) &blockCounter, arrSize * sizeof(int));
    hipMalloc((void**) &flags, arrSize * sizeof(int));

    ////Initialize all block counters to 0
    //for (unsigned int i = 0; i < arrSize; i++)
    //{
    //    blockCounter[i] = 0;
    //}

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    //Copy data to GPU
    startTime(&timer);
    
    //Copy data for the array from host to device
    hipMemcpy(arr_d, arr, arrSize * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    //Call kernel
    startTime(&timer);

    //Sorting on GPU
    if(arrSize > 1) 
	{
        //quicksort_advanced_kernel << < 1, 1, 0 >> > (arr_d, arrCopy, lessThan, greaterThan, partitionArr, lessThanSums, greaterThanSums, blockCounter, flags, 1, arrSize);
        quicksort_naive_kernel << < 1, 1, 0 >> > (arr_d, arrSize, 1);
    }

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time");

    //Copy data from GPU
    startTime(&timer);
    
    //After performing the quick sort, copy the sorted array from device to host
    hipMemcpy(arr, arr_d, arrSize * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    //Free GPU memory
    startTime(&timer);
    
    //Now that we are done, we can free the allocated memory to leave space for other computations
    hipFree(arr_d);
    hipFree(arrCopy);    
    hipFree(lessThan);
    hipFree(greaterThan);
    hipFree(lessThanSums);
    hipFree(greaterThanSums);
    hipFree(partitionArr);
    hipFree(blockCounter);
    hipFree(flags);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");
}