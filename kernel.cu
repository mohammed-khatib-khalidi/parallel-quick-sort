#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////
// Authors  : Lama Afra, Mohammad Al Khalidi, Taysseer Samman
// Usernames: laa59, mwa30, tjs00
// Course   : CMPS 396AA
// Timestamp: 20200328
// Project  : Parallel Quicksort
/////////////////////////////////////////////////////////////

#include "common.h"
#include "timer.h"

// CUDA Windows Headers
#if defined _WIN32 || defined _WIN64
#include "hip/hip_runtime.h"
#include ""
#endif

//The partition kernel method
__global__ void Partition_Kernel(float* arr, float* arrCopy, float* lessThan, float* greaterThan, int start, int end, int pivotIdx, int k)
{
	//Calculate the size of the array
    int arrSize = end - start + 1;
    
    //Compute the thread index
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    //Compute the real array index
    int index = start + tid;

	//In case the array was only one item then return the start index
	if (arrSize == 1) 
	{
        //Allow only the first thread to modify partition k
        if (tid == 0) 
			k = start;
        //Stop here
        return;
    }

    //Compute the pivot value
    float pivot = arr[pivotIdx];

    //Copy to temporary array
    arrCopy[tid] = arr[index];

    //Copy to the lessThan array
    if(arr[index] < pivot) 
        lessThan[tid] = 1;
    else 
        lessThan[tid] = 0;

    //Copy to the greaterThan array
    if(arr[index] > pivot) 
        greaterThan[tid] = 1;
    else
        greaterThan[tid] = 0;

    //Sync all threads
    __syncthreads();

    //Now we will start performing the prefix sum for the lessThan and greaterThan arrays
}

//Swap two elements of an array
__device__  void Swap_GPU(float* a, float* b)
{
	float temp = *a;
	*a = *b;
	*b = temp;
}

//Computes the partition after rearranging the array
__device__ int Partition_GPU(float* arr, int start, int end)
{
    //Index of smaller element
	int i = start - 1;

	for (int j = start; j < end; j++)
	{
		//If current element is smaller than the pivot
		if (arr[j] < arr[end])
		{
			//Increment the index of the smaller element
			i++;
			//Swap array elements with indices i and j
			Swap_GPU(&arr[i], &arr[j]);
		}
	}

	//Swap array elements with indices i + 1 and pivot
	Swap_GPU(&arr[i + 1], &arr[end]);

	//Return parition index
	return (i + 1);
}

//Naive version of the parallel quicksort which only parallelizes recursive calls
__global__ void Quicksort_Naive_Kernel(float* arr, int start, int end)
{
    //Partition
    int k = Partition_GPU(arr, start, end);

    if(start < k - 1) 
	{
        //Create cuda stream to run recursive calls in parallel
        hipStream_t s_left;

        //Set the non-blocking flag for the cuda stream
        hipStreamCreateWithFlags(&s_left, hipStreamNonBlocking);

        //Sort the left partition
		Quicksort_Naive_Kernel <<< 1, 1, 0, s_left >>> (arr, start, k - 1);

        //Destroy the stream after getting done from it
        hipStreamDestroy(s_left);
    }

    if(k + 1 < end) 
	{
        //Create cuda stream to run recursive calls in parallel
        hipStream_t s_right;

        //Set the non-blocking flag for the cuda stream
        hipStreamCreateWithFlags(&s_right, hipStreamNonBlocking);

        //Sort the right partition
		Quicksort_Naive_Kernel <<< 1, 1, 0, s_right >>> (arr, k + 1, end);

        //Destroy the stream after getting done from it
        hipStreamDestroy(s_right);
    }
}

//Advanced version of the parallel quicksort which parallelizes both the partition method and the recursive calls
__global__ void Quicksort_Advanced_Kernel(float* arr, int start, int end)
{
	//Get size of the array
	int arrSize = end - start + 1;

	//Allocate memory for the three arrays
	float* arrCopy;
	float* lessThan;
	float* greaterThan;
	hipMalloc((void**)&arrCopy, arrSize * sizeof(float));
	hipMalloc((void**)&lessThan, arrSize * sizeof(float));
	hipMalloc((void**)&lessThan, arrSize * sizeof(float));

	int pivotIdx = (start + end) / 2;

	//Configure the number of blocks and threads per block
	const unsigned int numThreadsPerBlock = 512;
	const unsigned int numBlocks = (arrSize + numThreadsPerBlock - 1) / numThreadsPerBlock;

	//Partition
	int k = 0;
	Partition_Kernel << < numBlocks, numThreadsPerBlock >> > (arr, arrCopy, lessThan, greaterThan, start, end, pivotIdx, k);

	//Sort the left partition
	if (start < k - 1) {
		Quicksort_Advanced_Kernel << < 1, 1 >> > (arr, start, k - 1);
	}

	//Sort the right partition
	if (k + 1 < end) {
		Quicksort_Advanced_Kernel << < 1, 1 >> > (arr, k + 1, end);
	}
}

void Quicksort_GPU(float* arr, int arrSize)
{
    //Define the timer
    Timer timer;

    //Allocate GPU memory
    startTime(&timer);
    
    //Declare and allocate the same array on the device
    float *arr_d;
    hipMalloc((void**) &arr_d, arrSize * sizeof(float));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    //Copy data to GPU
    startTime(&timer);
    
    //Copy data for the array from host to device
    hipMemcpy(arr_d, arr, arrSize * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    //Call kernel
    startTime(&timer);

    //Sorting on GPU
    if(arrSize > 1) 
	{
		Quicksort_Naive_Kernel << < 1, 1, 0 >> > (arr_d, 0, arrSize - 1);
    }

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time");

    //Copy data from GPU
    startTime(&timer);
    
    //After performing the quick sort, copy the sorted array from device to host
    hipMemcpy(arr, arr_d, arrSize * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    //Free GPU memory
    startTime(&timer);
    
    //Now that we are done, we can free the allocated memory to leave space for other computations
    hipFree(arr_d);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");
}