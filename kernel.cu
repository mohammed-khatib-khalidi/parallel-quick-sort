#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////
// Authors  : Lama Afra, Mohammad Al Khalidi, Taysseer Samman
// Usernames: laa59, mwa30, tjs00
// Course   : CMPS 396AA
// Timestamp: 20200328
// Project  : Parallel Quicksort
/////////////////////////////////////////////////////////////

#include "common.h"
#include "timer.h"

// CUDA Windows Headers
#if defined _WIN32 || defined _WIN64
#include "hip/hip_runtime.h"
#include ""
#endif

// The partition kernel method
__global__ void partition_kernel(float* arr, float* arrCopy, float* lessThan, float* greaterThan, int start, int end, int pivotIdx, int k)
{
	// Calculate the size of the array
    int arrSize = end - start + 1;
    
    // Compute the thread index
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Compute the real array index
    int index = start + tid;

	// In case the array was only one item then return the start index
	if (arrSize == 1) 
	{
        // Allow only the first thread to modify partition k
        if (tid == 0) 
			k = start;
        // Stop here
        return;
    }

    // Compute the pivot value
    float pivot = arr[pivotIdx];

    // Copy to temporary array
    arrCopy[tid] = arr[index];

    // Copy to the lessThan array
    if(arr[index] < pivot)
        lessThan[tid] = 1;
    else
        lessThan[tid] = 0;

    // Copy to the greaterThan array
    if(arr[index] > pivot) 
        greaterThan[tid] = 1;
    else
        greaterThan[tid] = 0;

    // Sync all threads
    __syncthreads();

    // Now we will start performing the prefix sum for the lessThan and greaterThan arrays
}

// Swap two elements of an array
__device__ void swap_gpu(float* a, float* b)
{
	float temp = *a;
	*a = *b;
	*b = temp;
}

// Computes the partition after rearranging the array
__device__ int partition_gpu(float* arr, int arrSize)
{
	// Index of smaller element
    int i = - 1;

	for (int j = 0; j < arrSize - 1; j++)
	{
		// If current element is smaller than the pivot
		if (arr[j] < arr[arrSize - 1])
		{
			// Increment the index of the smaller element
			i++;
			// Swap array elements with indices i and j
			swap_gpu(&arr[i], &arr[j]);
		}
	}

	// Swap array elements with indices i + 1 and pivot
	swap_gpu(&arr[i + 1], &arr[arrSize - 1]);

	// Return parition index
    return (i + 1);
}

// Naive version of the parallel quicksort which only parallelizes recursive calls
__global__ void quicksort_naive_kernel(float* arr, int arrSize)
{
    // Partition
    int k = partition_gpu(arr, arrSize);

    if(k > 1) 
	{
        // Create cuda stream to run recursive calls in parallel
        hipStream_t s_left;

        // Set the non-blocking flag for the cuda stream
        hipStreamCreateWithFlags(&s_left, hipStreamNonBlocking);

        // Sort the left partition
		quicksort_naive_kernel <<< 1, 1, 0, s_left >>> (&arr[0], k);

        // Destroy the stream after getting done from it
        hipStreamDestroy(s_left);
    }

    if(arrSize > k + 2) 
	{
        // Create cuda stream to run recursive calls in parallel
        hipStream_t s_right;

        // Set the non-blocking flag for the cuda stream
        hipStreamCreateWithFlags(&s_right, hipStreamNonBlocking);

        // Sort the right partition
		quicksort_naive_kernel <<< 1, 1, 0, s_right >>> (&arr[k + 1], arrSize - k - 1);

        // Destroy the stream after getting done from it
        hipStreamDestroy(s_right);
    }
}

//Advanced version of the parallel quicksort which parallelizes both the partition method and the recursive calls
__global__ void quicksort_advanced_kernel(float* arr, int start, int end)
{
	// Get size of the array
	int arrSize = end - start + 1;

	// Allocate memory for the three arrays
	float* arrCopy;
	float* lessThan;
	float* greaterThan;

	hipMalloc((void**)&arrCopy, arrSize * sizeof(float));
	hipMalloc((void**)&lessThan, arrSize * sizeof(float));
	hipMalloc((void**)&lessThan, arrSize * sizeof(float));

	int pivotIdx = (start + end) / 2;

	// Configure the number of blocks and threads per block
	const unsigned int numThreadsPerBlock = 512;
	const unsigned int numBlocks = (arrSize + numThreadsPerBlock - 1) / numThreadsPerBlock;

	// Partition
	int k = 0;
	partition_kernel << < numBlocks, numThreadsPerBlock >> > (arr, arrCopy, lessThan, greaterThan, start, end, pivotIdx, k);

	// Sort the left partition
	if (start < k - 1) 
	{
		quicksort_advanced_kernel << < 1, 1 >> > (arr, start, k - 1);
	}

	// Sort the right partition
	if (k + 1 < end) 
	{
		quicksort_advanced_kernel << < 1, 1 >> > (arr, k + 1, end);
	}
}

__host__ void quicksort_gpu(float* arr, int arrSize)
{
    //Define the timer
    Timer timer;

    //Allocate GPU memory
    startTime(&timer);
    
    //Declare and allocate required arrays on the device
    float* arr_d;
    float* arrCopy_d;
    float* lessThan_d;
    float* greaterThan_d;
    float* partition_d;
    hipMalloc((void**) &arr_d, arrSize * sizeof(float));
    hipMalloc((void**) &arrCopy_d, arrSize * sizeof(float));
    hipMalloc((void**) &lessThan_d, arrSize * sizeof(float));
    hipMalloc((void**) &greaterThan_d, arrSize * sizeof(float));
    hipMalloc((void**) &partition_d, arrSize * sizeof(float));

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    //Copy data to GPU
    startTime(&timer);
    
    //Copy data for the array from host to device
    hipMemcpy(arr_d, arr, arrSize * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    //Call kernel
    startTime(&timer);

    //Sorting on GPU
    if(arrSize > 1) 
	{
		quicksort_naive_kernel << < 1, 1, 0 >> > (arr_d, arrSize);
    }

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time");

    //Copy data from GPU
    startTime(&timer);
    
    //After performing the quick sort, copy the sorted array from device to host
    hipMemcpy(arr, arr_d, arrSize * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    //Free GPU memory
    startTime(&timer);
    
    //Now that we are done, we can free the allocated memory to leave space for other computations
    hipFree(arr_d);
    hipFree(arrCopy_d);
    hipFree(lessThan_d);
    hipFree(greaterThan_d);
    hipFree(partition_d);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");
}